#include "transMatrix.h"
#include "amgx_c.h"
#include <cassert>
//#include <sstream>

namespace Cuda {

void cudaSolver(CRS_ptr para_A, vec_ptr para_x, vec_ptr para_b)
{
    assert(para_A->innerSize() == para_b->size() && para_x->size() == para_b->size());

    AMGX_SAFE_CALL(AMGX_initialize());
    AMGX_SAFE_CALL(AMGX_initialize_plugins());

    /*
    std::stringstream conf_stream;
    conf_stream << "config_version=2, " << "solver(cg)=CG, " << "cg:preconditioner=NOSOLVER, " \
                << "cg:max_iters=1000, " <<"cg:tolerance=1e-6, " << "cg:monitor_residual=1, " << "cg:print_solve_stats=1";
    */

    AMGX_matrix_handle A;
    AMGX_vector_handle b;
    AMGX_vector_handle x;
    AMGX_resources_handle rsrc;
    AMGX_solver_handle solver;
    AMGX_config_handle cfg;

    //AMGX_config_create(&cfg, conf_stream.str().c_str());
    AMGX_config_create_from_file(&cfg, "E:\\AMGX_configs_xiao\\531config.json");
    AMGX_resources_create_simple(&rsrc, cfg);
    AMGX_matrix_create(&A, rsrc, AMGX_mode_dDDI);
    AMGX_vector_create(&b, rsrc, AMGX_mode_dDDI);
    AMGX_vector_create(&x, rsrc, AMGX_mode_dDDI);
    AMGX_solver_create(&solver, rsrc, AMGX_mode_dDDI, cfg);

    AMGX_pin_memory(para_A->outerIndexPtr(), sizeof(int) * (para_A->outerSize() + 1));
    AMGX_pin_memory(para_A->innerIndexPtr(), sizeof(int) * (para_A->nonZeros()));
    AMGX_pin_memory(para_A->valuePtr(), sizeof(ValueType) * (para_A->nonZeros()));
    AMGX_pin_memory(para_b->valuePtr(), sizeof(ValueType) * (para_b->size()));
    AMGX_pin_memory(para_x->valuePtr(), sizeof(ValueType) * (para_x->size()));

    AMGX_matrix_upload_all(A, para_A->outerSize(), para_A->nonZeros(), 1, 1, para_A->outerIndexPtr(), para_A->innerIndexPtr(), para_A->valuePtr(), NULL);
    AMGX_vector_upload(b, para_b->size(), 1, para_b->valuePtr());
    AMGX_vector_set_zero(x, para_x->size(), 1);

    AMGX_solver_setup(solver, A);
    AMGX_solver_solve_with_0_initial_guess(solver, b, x);

    AMGX_vector_download(x, para_x->valuePtr());

    AMGX_unpin_memory(para_A->outerIndexPtr());
    AMGX_unpin_memory(para_A->innerIndexPtr());
    AMGX_unpin_memory(para_A->valuePtr());
    AMGX_unpin_memory(para_b->valuePtr());
    AMGX_unpin_memory(para_x->valuePtr());

    AMGX_solver_destroy(solver);
    AMGX_vector_destroy(x);
    AMGX_vector_destroy(b);
    AMGX_matrix_destroy(A);
    AMGX_resources_destroy(rsrc);
    AMGX_SAFE_CALL(AMGX_config_destroy(cfg));

    AMGX_SAFE_CALL(AMGX_finalize_plugins());
    AMGX_SAFE_CALL(AMGX_finalize());

    return ;
}

}